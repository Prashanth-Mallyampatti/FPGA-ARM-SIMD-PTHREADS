/*

Name: Prashanth Mallyampatti
Student Id: 200250501
Unity Id: pmallya
Algorithm: Bitonic Sort

*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include<stdlib.h>
#include <time.h>
#include<assert.h>
#include<sys/time.h>

int threads_ = 0, blocks_ = 0;

#ifdef __cplusplus
extern "C"
{
#endif

__global__ 
void bitonic_sort(float *values, int j, int k)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int ij = i^j;

	//ascending
	if (ij > i && (i & k) == 0 && values[i] > values[ij])
	{
		float temp = values[i];
		values[i] = values[ij];
		values[ij] = temp;
	}

	//descending
	if (ij > i && (i & k) != 0 && values[i] < values[ij])
	{
		float temp = values[i];
		values[i] = values[ij];
		values[ij] = temp;
	}
}

int cuda_sort(int number_of_elements, float* a)
{
	//limiting thread usage
	if(number_of_elements > 512)
		threads_ = 512;
	else
		threads_ = number_of_elements;

	blocks_ = number_of_elements/threads_;
	
	float *values;
	size_t size = number_of_elements * sizeof(float);

	hipMalloc((void**) &values, size);
	hipMemcpy(values, a, size, hipMemcpyHostToDevice);

	dim3 threads(threads_, 1);
	dim3 blocks(blocks_, 1);

	for (int k = 2; k <= number_of_elements; k <<= 1) 
		for (int j=k>>1; j>0; j=j>>1) 
			bitonic_sort<<<blocks_, threads_>>>(values, j, k);
	
	hipMemcpy(a, values, size, hipMemcpyDeviceToHost);
	hipFree(values);

	return 0;
}



#ifdef __cplusplus
}
#endif

